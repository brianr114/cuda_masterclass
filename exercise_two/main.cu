﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void exercise_one()
{
    printf("BlockID x, y, z: %d, %d, %d | BlockDim x, y , z: %d, %d, %d | GridDim x, y, z: %d, %d, %d\n",
        blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

int main()
{
    const int N = 4;

    dim3 block(2, 2, 2);
    dim3 grid(N / block.x, N / block.y, N / block.z);

    exercise_one <<<grid, block>>> ();

    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}
